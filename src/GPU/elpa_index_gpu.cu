
#include <hip/hip_runtime.h>
extern "C" {
  int gpu_count() {
    int count;
    hipError_t cuerr = hipGetDeviceCount(&count);
    if (cuerr != hipSuccess) {
      count = -1000;
    }
    return count;
  }
}
